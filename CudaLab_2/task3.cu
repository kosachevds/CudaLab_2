#include "hip/hip_runtime.h"
#include "common.cuh"
#include <hip/hip_runtime.h>
#include <>
#include <iostream>

float multiplyOnCpu(unsigned const* const* left, unsigned const* const* right, unsigned* const* result, size_t size);
float multiplyGpuGlobal(unsigned const* const* left, unsigned const* const* right, unsigned* const* result, size_t size);
void copyMatrixToGpu(unsigned const* const* matrix, size_t size, unsigned* out_array);
void copyMatrixFromGpu(unsigned const* gpu_array, size_t one_dim_size, unsigned*const* matrix);
unsigned** createMatrix(size_t size);
unsigned** createRandomMatrix(size_t size);
void deleteMatrix(unsigned** matrix, size_t size);
void printMatrix(unsigned const* const* matrix, size_t size);

__global__ void multiplyGlobal(unsigned const* left, unsigned const* right, unsigned* result, size_t size)
{
    auto row = blockIdx.y * blockDim.y + threadIdx.y;
    auto col = blockIdx.x * blockDim.x + threadIdx.x;
    //int row = blockIdx.y * size + threadIdx.y;
    //int col = blockIdx.x * size + threadIdx.x;
    if (row < size && col < size) {
        auto sum = 0u;
        for (int k = 0; k < size; k++) {
            sum += left[row * size + k] * right[k * size + col];
        }
        result[row * size + col] = sum;
        //result[row * size + col] += left[row * size + col] + right[col * size + col];
    }
}

__global__ void kernel(unsigned const* a, unsigned const* b, unsigned* c, int size)
{
    const auto BLOCK_SIZE = 16;
    int bx = blockIdx.x; // ������� �����
    int by = blockIdx.y; //
    int tx = threadIdx.x; // ������� ���� ������ �����
    int ty = threadIdx.y; //
    int aBegin = size * BLOCK_SIZE * by;
    int aEnd = aBegin + size - 1;
    int aStep = BLOCK_SIZE;
    int bBegin = bx * BLOCK_SIZE;
    int bStep = BLOCK_SIZE * size;
    float sum = 0.0f;
    for (int ia = aBegin, ib = bBegin; ia <= aEnd; ia += aStep, ib += bStep) {
        __shared__ float as [BLOCK_SIZE][BLOCK_SIZE];
        __shared__ float bs [BLOCK_SIZE][BLOCK_SIZE];
        as[ty][tx] = a[ia + size * ty + tx];
        bs[ty][tx] = b[ib + size * ty + tx];
        __syncthreads(); // ��������, ��� ���������� ��������� ���������
        for (int k = 0; k < BLOCK_SIZE; k++)
            sum += as[ty][k] * bs[k][tx];
        __syncthreads(); // ��������, ��� ���������� ������ ������ �� �����
    }
    c[size * BLOCK_SIZE * by + BLOCK_SIZE * bx + size * ty + tx] = sum;
}


void task3()
{
    const auto SIZE = 5;
    auto left = createRandomMatrix(SIZE);
    printMatrix(left, SIZE);
    std::cout << std::endl;
    auto right = createRandomMatrix(SIZE);
    printMatrix(right, SIZE);
    std::cout << std::endl;
    auto result = createMatrix(SIZE);
    //multiplyOnCpu(left, right, result, SIZE);
    multiplyGpuGlobal(left, right, result, SIZE);
    printMatrix(result, SIZE);
    std::cout << std::endl;
    deleteMatrix(left, SIZE);
    deleteMatrix(right, SIZE);
    deleteMatrix(result, SIZE);
}

float multiplyOnCpu(unsigned const* const* left, unsigned const* const* right, unsigned* const* result, size_t size)
{
    hipEvent_t start, end;
    hipEventCreate(&start);
    hipEventCreate(&end);
    hipEventRecord(start);
    for (auto i = 0u; i < size; ++i) {
        for (auto j = 0; j < size; ++j) {
            result[i][j] = 0u;
            for (auto k = 0; k < size; ++k) {
                result[i][j] += left[i][k] * right[k][j];
            }
        }
    }
    hipEventRecord(end);
    hipEventSynchronize(end);
    float ms;
    hipEventElapsedTime(&ms, start, end);
    return ms;
}

float multiplyGpuGlobal(unsigned const* const* left, unsigned const* const* right, unsigned* const* result, size_t size)
{
    const auto SIZE = size * size;
    unsigned *left_gpu, *right_gpu, *result_gpu;
    hipMalloc(reinterpret_cast<void**>(&left_gpu), SIZE * sizeof(unsigned));
    copyMatrixToGpu(left, size, left_gpu);
    hipMalloc(reinterpret_cast<void**>(&right_gpu), SIZE * sizeof(unsigned));
    copyMatrixToGpu(right, size, right_gpu);
    hipMalloc(reinterpret_cast<void**>(&result_gpu), SIZE * sizeof(unsigned));

    hipEvent_t start, end;
    hipEventCreate(&start);
    hipEventCreate(&end);
    unsigned block_count, block_size;
    if (SIZE <= MAX_BLOCK_SIZE) {
        block_size = SIZE;
        block_count = 1;
    } else {
        block_size = MAX_BLOCK_SIZE;
        block_count = SIZE / MAX_BLOCK_SIZE + 1;
    }
    hipEventRecord(start);
    multiplyGlobal<<<1, dim3(size, size)>>>(left_gpu, right_gpu, result_gpu, size);
    hipEventRecord(end);
    hipEventSynchronize(end);
    float ms;
    hipEventElapsedTime(&ms, start, end);
    copyMatrixFromGpu(result_gpu, size, result);

    hipEventDestroy(start);
    hipEventDestroy(end);
    hipFree(left_gpu);
    hipFree(right_gpu);
    hipFree(result_gpu);
    return ms;
}

void copyMatrixToGpu(unsigned const* const* matrix, size_t size, unsigned* out_array)
{
    std::vector<unsigned> buffer;
    buffer.reserve(size * size);
    for (int i = 0; i < size; ++i) {
        for (int j = 0; j < size; ++j) {
            buffer.push_back(matrix[i][j]);
        }
    }
    hipMemcpy(out_array, buffer.data(), buffer.size() * sizeof(unsigned), hipMemcpyHostToDevice);
}

void copyMatrixFromGpu(unsigned const* gpu_array, size_t one_dim_size, unsigned*const* matrix)
{
    std::vector<unsigned> buffer(one_dim_size * one_dim_size);
    hipMemcpy(buffer.data(), gpu_array, buffer.size() * sizeof(unsigned), hipMemcpyDeviceToHost);
    auto index = 0;
    for (int i = 0; i < one_dim_size; ++i) {
        for (int j = 0; j < one_dim_size; ++j) {
            matrix[i][j] = buffer[index];
            ++index;
        }
    }
}

unsigned** createMatrix(size_t size)
{
    auto matrix = new unsigned*[size];
    for (auto i = 0; i < size; ++i) {
        matrix[i] = new unsigned[size];
    }
    return matrix;
}

unsigned** createRandomMatrix(size_t size)
{
    auto matrix = createMatrix(size);
    for (auto i = 0; i < size; ++i) {
        for (auto j = 0; j < size; ++j) {
            matrix[i][j] = rand() % 100 + 1;
        }
    }
    return matrix;
}

void deleteMatrix(unsigned** matrix, size_t size)
{
    for (auto i = 0; i < size; ++i) {
        delete[] matrix[i];
    }
    delete[] matrix;
}

void printMatrix(unsigned const* const* matrix, size_t size)
{
    for (auto i = 0; i < size; ++i) {
        for (auto j = 0; j < size; ++j) {
            std::cout << matrix[i][j] << ' ';
        }
        std::cout << std::endl;
    }
}
