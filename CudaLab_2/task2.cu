#include "hip/hip_runtime.h"
#include "common.cuh"
#include <hip/hip_runtime.h>
#include <>
#include <fstream>
#include <ctime>

__global__ void task2Kernel(unsigned const* a, unsigned const* b, unsigned* result, size_t size)
{
    auto index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index >= size) {
        return;
    }
    result[index] = a[index] * b[index];
}

void task2()
{
    const auto SIZE = 8 * 1024 * 1024;
    const auto STREAM_COUNT = 8;
    srand(time(nullptr));
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    // TODO: matrix[3][SIZE];
    std::vector<unsigned> host_a(SIZE), host_b(SIZE), host_c(SIZE);
    unsigned *dev_a, *dev_b, *dev_c;
    hipMalloc(reinterpret_cast<void**>(&dev_a), SIZE * sizeof(unsigned));
    hipMalloc(reinterpret_cast<void**>(&dev_b), SIZE * sizeof(unsigned));
    hipMalloc(reinterpret_cast<void**>(&dev_c), SIZE * sizeof(unsigned));
    std::vector<float> times;
    hipStream_t streams[STREAM_COUNT];
    for (auto count = 1; count <= STREAM_COUNT; ++count) {
        for (auto i = 0; i < count; ++i) {
            hipStreamCreate(&streams[i]);
        }
        for (auto i = 0; i < SIZE; ++i) {
            host_a[i] = rand() % 1024 + 1;
            host_b[i] = rand() % 1024 + 1;
        }
        auto chunk_size = SIZE / count;
        //hipEventRecord(start);
        for (auto i = 0; i < count; ++i) {
            auto begin = i * chunk_size;
            int size;
            if (i != count - 1) {
                size = SIZE - begin;
            } else {
                size = chunk_size;
            }
            auto bytes = size * sizeof(unsigned);
            hipEventRecord(start, streams[i]);
            hipMemcpyAsync(dev_a + begin, host_a.data() + begin, bytes, hipMemcpyHostToDevice, streams[i]);
            hipMemcpyAsync(dev_b + begin, host_b.data() + begin, bytes, hipMemcpyHostToDevice, streams[i]);
            task2Kernel<<<SIZE / MAX_BLOCK_SIZE, MAX_BLOCK_SIZE, 0, streams[i]>>>(
                dev_a + begin, dev_b + begin, dev_c + begin, size);
            hipMemcpyAsync(host_c.data() + begin, dev_c + begin, bytes, hipMemcpyDeviceToHost, streams[i]);
            hipDeviceSynchronize();
            hipEventRecord(stop, streams[i]);
        //hipEventSynchronize(stop);
        }
        //hipEventRecord(stop);
        //hipEventSynchronize(stop);
        for (auto i = 0; i < count; ++i) {
            hipStreamDestroy(streams[i]);
        }
        float ms;
        hipEventElapsedTime(&ms, start, stop);
        times.push_back(ms);
    }
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    std::ofstream out("times.txt");
    writeVector(times, out);
    out.close();
}